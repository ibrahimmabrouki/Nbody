#include "hip/hip_runtime.h"
Skip to content
Navigation Menu
hamdanabdellatef
/
Nbody

Type / to search
Code
Issues
Pull requests
Actions
Projects
Security
Insights
 The password you provided is in a list of passwords commonly used on other websites. To increase your security, you must update your password. After December 19, 2024 we will automatically reset your password. Change your password on the settings page.

Read our documentation on safer password practices.

Comparing changes
Choose two branches to see what’s changed or to start a new pull request. If you need to, you can also  or learn more about diff comparisons.
 
 
...
 
 
  Able to merge. These branches can be automatically merged.
Discuss and review the changes in this comparison with others. Learn about pull requests
 1 commit
 1 file changed
 1 contributor
Commits on Nov 19, 2024
Create optimizednbody.cu

@ibrahimmabrouki
ibrahimmabrouki authored 10 minutes ago
 Showing  with 118 additions and 0 deletions.
 118 changes: 118 additions & 0 deletions118  
optimizednbody.cu
Original file line number	Original file line	Diff line number	Diff line change
@@ -0,0 +1,118 @@
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 256  // Number of threads per block

typedef struct { float x, y, z, vx, vy, vz; } Body;

/* Device kernel to calculate gravitational forces between bodies*/

_global_ void bodyForceKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;  // Ensure thread doesn't access out-of-bounds memory

    float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

    for (int j = 0; j < n; j++) {
        float dx = p[j].x - p[i].x;
        float dy = p[j].y - p[i].y;
        float dz = p[j].z - p[i].z;
        float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3;
        Fy += dy * invDist3;
        Fz += dz * invDist3;
    }

    p[i].vx += dt * Fx;
    p[i].vy += dt * Fy;
    p[i].vz += dt * Fz;
}

/* Device kernel to integrate positions of bodies.*/

_global_ void integratePositionsKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;  // Ensure thread doesn't access out-of-bounds memory

    p[i].x += p[i].vx * dt;
    p[i].y += p[i].vy * dt;
    p[i].z += p[i].vz * dt;
}

int main(const int argc, const char** argv) {
    int nBodies = 2 << 11;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    const char *initialized_values;
    const char *solution_values;

    if (nBodies == 2 << 11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f;  // Time step
    const int nIters = 10;   // Simulation iterations

    int bytes = nBodies * sizeof(Body);
    Body p = (Body)malloc(bytes);
    Body *d_p;

    // Read initial values from file
    read_values_from_file(initialized_values, (float*)p, bytes);

    // Allocate memory on the GPU
    hipMalloc(&d_p, bytes);

    // Copy initial values to GPU
    hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

    double totalTime = 0.0;

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        // Launch the bodyForce kernel
        int blocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bodyForceKernel<<<blocks, BLOCK_SIZE>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();

        // Launch the integratePositions kernel
        integratePositionsKernel<<<blocks, BLOCK_SIZE>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)nIters;
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

    // Copy results back to CPU
    hipMemcpy(p, d_p, bytes, hipMemcpyDeviceToHost);

    // Write results to file
    write_values_to_file(solution_values, (float*)p, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    // Free allocated memory
    free(p);
    hipFree(d_p);

    return 0;
}
Footer
© 2024 GitHub, Inc.
Footer navigation
Terms
Privacy
Security
Status
Docs
Contact
Manage cookies
Do not share my personal information
